#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main() {
    int devicesCount;
    if(hipSuccess != hipGetDeviceCount( &devicesCount ))
        return std::printf("GetDeviceCount failed.");

    for (unsigned i = 0; i < devicesCount; ++i) {
        if(hipSuccess != hipSetDevice(i))
            return std::printf("SetDevice failed.");

        int freeMemory, totalMemory;
        if(hipSuccess != hipMemGetInfo(&freeMemory, &totalMemory))
            return std::printf("MemGetInfo failed.");

        hipDeviceProp_t properties;
        if(hipSuccess != hipGetDeviceProperties(&properties, i))
            return std::printf("GetDeviceProperties failed.");

        std::cout << "Device: " << properties.name << ". Free memory = " << freeMemory << " bytes, total memory = " <<
                  totalMemory << " bytes.\n" << "Max threads per block: " << properties.maxThreadsPerBlock << ", max threads dimension: (" <<
                  properties.maxThreadsDim[0] << ", " << properties.maxThreadsDim[1] << ", " << properties.maxThreadsDim[2] <<
                  "), max grid size: (" << properties.maxGridSize[0] << ", " << properties.maxGridSize[1] << ", " <<
                  properties.maxGridSize[2] << ").\nTotal global memory: " << properties.totalGlobalMem << " bytes, total const memory: " <<
                  properties.totalConstMem << " bytes, memory bus width: " << properties.memoryBusWidth << " bits, l2 cache size: " <<
                  properties.l2CacheSize << " bytes.\n\n";
    }
    return 0;
}